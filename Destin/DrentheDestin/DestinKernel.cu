#include "hip/hip_runtime.h"
#include "DestinKernel.h"

// C/C++ headers
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sstream>
#include <stdexcept>

// Cuda header
#include <hip/hip_runtime.h>
#include <hiprand.h>

const int AmountThreads = 128;


using namespace std;

__global__ void CalculateDistance( int States, int InputDimensionlity, float *InputData, float *CentroidVectorData, float *CentroidDist, float *CentroidStarvation);
__global__ void CalculateWinningCentroids( int States, float *CentroidDist, int *WinningCentroids );
__global__ void UpdateStarvation( int States, float StarvationCoefficient, int *WinningCentroids, float *CentroidStarvation );
__global__ void UpdateWinningCentroids( int States, int InputDimensionlity, float LearningRate, float *InputData, float *CentroidVectorData, int *WinningCentroids, float *CentroidDist );
__global__ void CalculatePOS( int States, float *CentroidDist, float *Output );
__global__ void UpdateBeliefs( const int states, float *dPOS, float * dNewBeliefs, float * dOldBeliefs, int * dCountingTables,	int * dParentsInputAdvice, int parentStates, int * dSumTables, bool isPSSATraining, int * dOutputAdvice);
__device__ void find_max(const int thread_start, const int threadcount, const int length,float * winner, int * winnerId );
__device__ void updateCountingTables(int mStates, int parentStates, int * dCountingTables, int advice, int previousWinningBelief, int newWinningBelief, int * dSumTables, int bid);
__global__ void initializeMemory( const int states, const int parentStates, float * dBeliefs, int * dOutputAdvice, int * dCountingTables, int * dSumTables );

DestinKernel::DestinKernel( void )
{
    mID=0;
	mRows=0;
	mCols=0;
	mStates=0;
	mParentStates=0;
	mInputDimensionlity=0;
    mLearningRate = 0;
    mSTARVATION_COEFFICIENT = 0;
	hipGetDeviceCount(&mDevices);
	dParentInputAdvice=NULL;
	mIsCentroidsTraining = true;
	mIsPSSATraining = true;
	cout << "Kernel created" << endl;
}

void DestinKernel::free(){
	//everything is set to null so in case free is called twice,
	//then it wont try to delete old dangling pointers.
    hipFree( dCentroidsVectorData ); dCentroidsVectorData = NULL;
    hipFree( dPOS ); dPOS = NULL;
    hipFree( dCentroidsDistance ); dCentroidsDistance = NULL;
    hipFree( dCentroidStarvation ); dCentroidStarvation  = NULL;
    hipFree( dWinningCentroids ); dWinningCentroids = NULL;
    hipFree( dBeliefs );dBeliefs = NULL;
    hipFree( dOutputAdvice ); dOutputAdvice = NULL;
    hipFree( dCountingTables );dCountingTables = NULL;
    hipFree( dSumTables );dSumTables = NULL;

    delete [] mCentroidsDistance;mCentroidsDistance = NULL;
    delete [] mCentroidStarvation;mCentroidStarvation = NULL;
    delete [] mWinningCentroids;mWinningCentroids = NULL;
    delete [] mPOS;mPOS = NULL;
    delete [] mCentroidWinCounter;mCentroidWinCounter = NULL;
    delete [] mBeliefs;mBeliefs = NULL;

    cout << "Kernel destroyed" << endl;
}

DestinKernel::~DestinKernel( void )
{
	this->free();
}

#define CUDA_TEST_MALLOC( p, s )                                                                           \
	if( hipMalloc( p , s ) != 0 ){                                                                        \
		 stringstream mess; mess << "could not cudaMaclloc at " << __FILE__ << ":" << __LINE__ << endl ;   \
         mess << "trying to allocate with size " << s << endl ;            		             		       \
	     throw runtime_error(mess.str());}                                                                 \

void DestinKernel::Create( int ID, int Rows, int Cols, int States, int ParentStates, int InputDimensionlity, float FixedLeaningRate, hiprandGenerator_t gen)
{
    mID = ID;
    mRows = Rows;
    mCols = Cols;
    mStates = States;
    mParentStates = ParentStates;
    mInputDimensionlity = InputDimensionlity;
    mLearningRate = FixedLeaningRate;
    if(ParentStates==0){
	throw logic_error("ParentStates must be at least one. Set = 1 if this is the top layer");
    }
    mSTARVATION_COEFFICIENT = 1.0/((float)InputDimensionlity*(float)InputDimensionlity);
    if ( mSTARVATION_COEFFICIENT < 1.0/512.0 )
    {
        mSTARVATION_COEFFICIENT=1.0/512.0;
    }

    // Define the data sizes
    // Size of de nodes is rows times columns
    sizeOfNodes = mRows*mCols;
    // Size of the data of nodes is rows times columns times centroids
    sizeOfNodeData = sizeOfNodes*mStates;

    // Size of the layer with all vectors is rows times columns times centroids times the input (also observation) vector length.
    sizeOfLayerData = sizeOfNodeData*mInputDimensionlity;
    // Keep track of which centroid won
    mCentroidWinCounter = new int[sizeOfNodeData];
    for(int c=0;c<sizeOfNodeData;c++)
    {
        mCentroidWinCounter[c] = 0;
    }

    //TODO: put in error checking incase the cudaMallocs fail in case of not enough memory on device
    // Array full with all the winning centroids of each node
    mWinningCentroids = new int[sizeOfNodes];
    CUDA_TEST_MALLOC( (void**)&dWinningCentroids, sizeOfNodes*sizeof(int) );

    // Node data contains the distance to the observation of all centroids (It's is empty the first run)
    mCentroidsDistance = new float[sizeOfNodeData];
    CUDA_TEST_MALLOC( (void**)&dCentroidsDistance, sizeOfNodeData*sizeof(float) );

    // Starvation data for all centroids
    mCentroidStarvation = new float[sizeOfNodeData];
    CUDA_TEST_MALLOC( (void**)&dCentroidStarvation, sizeOfNodeData*sizeof(float) );
    for(int i=0;i<sizeOfNodeData;i++)
    {
        mCentroidStarvation[i]=1.0f;
    }
    // Copy the data from host to device
    hipMemcpy(dCentroidStarvation, mCentroidStarvation, sizeOfNodeData*sizeof(float), hipMemcpyHostToDevice);

    //TODO: make sure this POS is being fed to the correct place, and if it needs to go back to the host
    //POS - P(o|s') of update equation
    mPOS = new float[sizeOfNodeData];
    CUDA_TEST_MALLOC( (void**)&dPOS, sizeOfNodeData*sizeof(float) );

    CUDA_TEST_MALLOC( (void**)&dCentroidsVectorData, sizeOfLayerData*sizeof(float) );

    // This is to fill the dLayerData with all random numbers between 0.0 and 1.0
    hiprandGenerateUniform( gen, dCentroidsVectorData, sizeOfLayerData );

    //Node belief output, fed as input to parent nodes
    mBeliefs = new float[sizeOfNodeData];
    CUDA_TEST_MALLOC((void**)&dBeliefs, sizeOfNodeData * sizeof(float));
    
    //Node advice for fed to child nodes
    CUDA_TEST_MALLOC((void**)&dOutputAdvice, sizeOfNodes * sizeof(int));

    //Used in P(s'|s,a) calculations, counts when node transitions from s to s' when parent advice = a
    cout << "mRows " << mRows <<  "mCols " << mCols << "mParentStates " << mParentStates << " mStates " << mStates
    		<< " sizeof int " << sizeof(int) << endl ;
    CUDA_TEST_MALLOC((void**)&dCountingTables, mRows * mCols * mParentStates * mStates * mStates * sizeof(int));

    //Used in P(s'|s,a) (aka PSSA) calculations, holds the sum of the counting table columns
    CUDA_TEST_MALLOC((void**)&dSumTables, mRows * mCols * mParentStates * mStates * sizeof(int));

    dim3 grid(mCols, mRows); //grid of nodes (aka blocks)
    dim3 threads(AmountThreads);//threads per node

    //initialize memory to uniform distribution
    initializeMemory<<<grid, threads>>>(mStates, mParentStates, dBeliefs, dOutputAdvice, dCountingTables, dSumTables);
}

__global__ void initializeMemory( const int states, const int parentStates, float * dBeliefs, int * dOutputAdvice, int * dCountingTables, int * dSumTables ){
	int bid = blockIdx.x + blockIdx.y * gridDim.x;
	int tid = threadIdx.x;
	//dBeliefs[bid]
	int threads = blockDim.x ;

	const float uniform_c = 1.0 / (float)states;
	for(int t = tid ; t < states ; t+=threads ){
		dBeliefs[bid*states+t] = uniform_c;
	}

	if(tid==0){
		dOutputAdvice[bid] = 0;
	}

	//Each node has N=parentStates counting tables, each is size states x states
	for(int t = tid ; t < parentStates * states * states ; t+=threads){
		dCountingTables[bid * parentStates * states * states + t] = 1;
	}
	//holds the sums of the columns of the counting tables
	for(int t = tid ; t < parentStates * states ; t+=threads){
		dSumTables[bid*parentStates * states + t ] = states;
	}
}

void DestinKernel::DoDestin( float *Input, stringstream * xml )
{
    // Threads is the amount of thread inside each block
    dim3 threads( AmountThreads );
    // Grid is the amount of blocks inside a grid.
    dim3 grid( mCols, mRows );
    // Cause of the use of dynamic shared memory you have to tell the kernel how much shared memory space you need for each block.
    int sharedMem;
    // The launch of the kernels itself with centroids(states), dimension, input data and the Data of the layer itself
    // Calculating the distance of the centroids to an observation
    sharedMem = (mInputDimensionlity+mInputDimensionlity)*sizeof(float);
    CalculateDistance<<<grid, threads, sharedMem>>>( mStates, mInputDimensionlity, Input, dCentroidsVectorData, dCentroidsDistance, dCentroidStarvation );

    if(mIsCentroidsTraining){
		sharedMem = (mStates+mStates)*sizeof(float);
		// Kernel for finding the winning centroids
		CalculateWinningCentroids<<<grid, threads, sharedMem>>>( mStates, dCentroidsDistance, dWinningCentroids );
		// Kernel for starvation updates
		UpdateStarvation<<<grid, threads>>>( mStates, mSTARVATION_COEFFICIENT, dWinningCentroids, dCentroidStarvation );
		// Kernel for updating winning centroids
		sharedMem = mInputDimensionlity*sizeof(float);
		UpdateWinningCentroids<<<grid, threads, sharedMem>>>( mStates, mInputDimensionlity, mLearningRate, Input, dCentroidsVectorData, dWinningCentroids, dCentroidsDistance );
    }

    // Kernel for calculating P(o|s')
    sharedMem = (mStates+mStates)*sizeof(float);
    CalculatePOS<<<grid, threads, sharedMem>>>( mStates, dCentroidsDistance, dPOS );

    //TODO: rename dNewBeliefs and dOldBeliefs to just dBeliefs, get rid of one.
    int n = mStates > 16 ? 16 : mStates ;
    dim3 threads_plane( n, n ); 
    //total threads should be less than 512 per block, hardware limit so states needs to be less<=22
    //Chose 16 because seems like it would play better than 22... but not sure.
    //TODO: this should probably be a multiple of states instead to avoid wasting threads
    //TODO: might make sense to break up the UpdateBeliefs kernel because a lot is done with just a single row of threads so
    //lots of the threads are wasted, not sure if this would outweigh the overhead of a separate kernel launches
    sharedMem = (mStates * mStates + mStates) * sizeof(float);

    UpdateBeliefs<<<grid, threads_plane, sharedMem >>>(mStates, dPOS, dBeliefs, dBeliefs, dCountingTables, dParentInputAdvice, mParentStates, dSumTables, mIsPSSATraining, dOutputAdvice) ;
    

    this->WriteData(xml);
}

void DestinKernel::WriteData( stringstream * xmls )
{
    hipMemcpy(mCentroidsDistance, dCentroidsDistance, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mCentroidStarvation, dCentroidStarvation, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mPOS, dPOS, sizeOfNodeData*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mWinningCentroids, dWinningCentroids, sizeOfNodes*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(mBeliefs, dBeliefs, sizeOfNodeData * sizeof(float), hipMemcpyDeviceToHost);

    if(xmls!=NULL){
    	stringstream & xml(*xmls);
        xml << "<layer id=\"" << mID << "\">" << endl;
        for(int r=0;r<mRows;r++)
        {
            for(int c=0;c<mCols;c++)
            {
                int winningCentroid = mWinningCentroids[r*mCols+c];
                // winning counter finds place on the host might not be the best place to put this still
                // cause we are already writing here some output why create a special loop for it.
                mCentroidWinCounter[(c+r*mCols)*mStates+winningCentroid] += 1;
                xml << "<node id=\"" << r*mCols+c << "\" centroidWin=\"" << mWinningCentroids[r*mCols+c] << "\">" << endl;
                for(int s=0;s<mStates;s++)
                {
                    xml << "<centroid id=\"" << s << "\" ";
                    xml << "lastDistance=\"" << mCentroidsDistance[(c+r*mCols)*mStates+s] << "\" ";
                    xml << "starvation=\"" << mCentroidStarvation[(c+r*mCols)*mStates+s] << "\" ";
                    xml << "POS=\"" << mPOS[(c+r*mCols)*mStates+s]  << "\" ";
                    xml << "winCount=\"" << mCentroidWinCounter[(c+r*mCols)*mStates+s]  << "\"";
                    xml << "/>" << endl;
                }
                xml << "</node>" << endl;
            }
        }
        xml << "</layer>" << endl;
    }

}
// ***********************
// DeSTIN inside CUDA Part
// ***********************
__global__ void CalculateDistance( int States, int InputDimensionlity, float *InputData, float *CentroidVectorData, float *CentroidDist, float *CentroidStarvation)
{
    // This is how to declare a shared memory inside CUDA.
    extern __shared__ float shared[];
    float* input = (float*)&shared;
    float* distance = (float*)&input[InputDimensionlity];

    // We use many threads they need to know where they have to do there work.
    // tid (Thread ID) is the amount of threads inside a block its a fixed amount it can be changed by changing: AmountThreads.
    // Keep in mind that CUDA threads should be in steps of 32 (each warp takes 4 clock cycles where each cycle calculate 8 threads)
    int tid = threadIdx.x;
    // bid (Block ID) this keeps track in which  node we are working you can ask the grid the size of the blocks used in x or y and on a Fermi or higher even z
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    // make sure the input data is inside shared memory this we are going to compare the amount of centroids.
    while(tid < InputDimensionlity)
    {
        // Put input data for node inside shared memory
        input[tid] = InputData[tid + bid * InputDimensionlity];
        // A trick for when the dimension is bigger then the amount of threads
        tid += blockDim.x;
    }
    // all threads have to be here to be sure shared memory is filled with the input.
    __syncthreads();

    // calculation distance in massive thread style.
    // keep track of the centroid
    int centroid = 0;
    //TODO: should be able to parallelize over states
    while (centroid<States)
    {
        // reset the tid
        tid = threadIdx.x;
        while(tid < InputDimensionlity)
        {
            // This temp will have for a short while the calculation of input - centroid for position tid (one cell of the vector)
            float temp = 0.0f;
            // distance to input = (input - centroid)*(input - centroid)
            // Small formula to get to the right working position: dimension*centroids*block+current centroid*dimension+thread
            temp = input[tid] - CentroidVectorData[InputDimensionlity*States*bid+centroid*InputDimensionlity+tid] ;
            distance[tid] = temp * temp;
            // A trick for when the dimension is bigger then the amount of threads
            tid += blockDim.x;
        }
        // all threads have to wait here so we know all distance have been calculated
        __syncthreads();

        // Cause DeSTIN don't work with numbers that are 2^? we have to check for odd numbers
        int dOld = InputDimensionlity;
        // bite wise divide by 2 (should be faster the /2)
        int d = InputDimensionlity >> 1;
        // a sum reduction, This is a common trick on CUDA to add shared memory instead of striding true memory
        // You have to use half the memory each step and each thread will add itself to with the other half.
        while (d != 0)
        {
            // reset the tid
            tid = threadIdx.x;
            dOld = dOld - d*2;
            while(tid < d)
            {
                // the adding calculation
                distance[tid] += distance[tid + d];

                // special case in case of odd number (As long as this doesn't happen too often it won't effect speed)
                if (dOld == 1 && tid == d-1)
                {
                    distance[tid] += distance[tid + d + 1];
                }
                tid += blockDim.x;
            }
            // Sync moment before starting with next iteration of reduction.
            __syncthreads();

            dOld = d;
            d >>= 1;
        }

        // Write distance to Node Data
        tid = threadIdx.x;
        if(tid == 0)
        {
            // square root on sum of the (input - centroid)*(input - centroid)
            // (Remember that you should copy the data from the device to the host and store it then)
            CentroidDist[centroid+bid*States] = (sqrt(distance[tid]))*CentroidStarvation[centroid+bid*States];
        }
        // go to next centroid inside the node (bid is taking care of the other node)
        centroid++;
    }
}

	// To reduce the amount of work that one kernel is doing i have decided that splitting the work over more kernels should speed up the whole procces
   //TODO: make sure this can work with __shared__ as is
__global__ void CalculateWinningCentroids( int States, float *CentroidDist, int *WinningCentroids )
{
    extern __shared__ float shared[];
    float* winner = (float*)&shared;
    float* winnerId = (float*)&winner[States];
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    while(tid < States)
    {
        winnerId[tid] = tid;
        winner[tid] = CentroidDist[tid+bid*States];
        tid += blockDim.x;
    }
    __syncthreads();

    int dOld = States;
    int d = States >> 1;
    while (d != 0)
    {
        tid = threadIdx.x;
        dOld = dOld - d*2;
        while(tid < d)
        {
            if(winner[tid] > winner[tid + d])
            {
                // Move winning centroid to the beginning
                winner[tid] = winner[tid + d];
                winnerId[tid] = winnerId[tid + d];
            }

            if (dOld == 1 && tid == d-1)
            {
                // special case of odd numbers
                if(winner[tid] > winner[tid + d + 1])
                {
                    winner[tid] = winner[tid + d + 1];
                    winnerId[tid] = winnerId[tid + d + 1];
                }
            }
            tid += blockDim.x;
        }
        // Sync moment before starting with next iteration of reduction.
        __syncthreads();

        dOld = d;
        d >>= 1;
    }
    // Write the winning centroid into there position
    tid = threadIdx.x;
    if(tid == 0)
    {
        WinningCentroids[bid] = winnerId[tid];
    }
}

// This is the updating starvation fast and quick to update all the nodes and reset the winning centroid
// According to DeSTIN paper: The winning centroid starvation gets reset while the others starve more
// Aldo this is the simple version of it it might be changed in the further cause this make the network also forget what it learn
// when it is looking at something else for a very long time (Short and Long term memory)
__global__ void UpdateStarvation( int States, float StarvationCoefficient, int *WinningCentroids, float *CentroidStarvation )
{
    // for tid and bid see CalculateDistance kernel.
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;
    while(tid < States)
    {
        // Let all centroid starve
        CentroidStarvation[tid+bid*States] = (1.0f-StarvationCoefficient)*CentroidStarvation[tid+bid*States];
        // Reset winning centroid
        CentroidStarvation[WinningCentroids[bid]+bid*States] = 1.0f;
        tid += blockDim.x;
    }
}

// Move the winning centroids closer to the observation
__global__ void UpdateWinningCentroids( int States, int InputDimensionlity, float LearningRate, float *InputData, float *CentroidVectorData, int *WinningCentroids, float *CentroidDist )
{
    extern __shared__ float newDistance[];
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    int centroid = WinningCentroids[bid];
    float temp;
    float inputD;
    int pos;

    //this while block calculates the distance between the input vectors
    //and the centroid vectors
    while(tid < InputDimensionlity)
    {
        pos = InputDimensionlity*States*bid+centroid*InputDimensionlity+tid;
        temp = CentroidVectorData[pos];
        inputD = InputData[tid + bid * InputDimensionlity];
        temp = inputD - (temp * LearningRate);
        CentroidVectorData[pos] = temp;
        temp = (inputD - temp) * (inputD - temp);
        newDistance[tid] = temp;

        tid += blockDim.x;
        pos += blockDim.x;
    }
    __syncthreads();

    int dOld = InputDimensionlity;
    int d = InputDimensionlity >> 1;
    while (d != 0)
    {
        tid = threadIdx.x;
        dOld = dOld - d*2;
        while(tid < d)
        {
            newDistance[tid] += newDistance[tid + d];
            if (dOld == 1 && tid == d-1)
            {
                // special case of odd numbers
                newDistance[tid] += newDistance[tid + d + 1];
            }
            tid += blockDim.x;
        }
        // Sync moment before starting with next iteration of reduction.
        __syncthreads();

        dOld = d;
        d >>= 1;
    }

    tid = threadIdx.x;
    if(tid == 0)
    {
        CentroidDist[centroid+bid*States] = sqrt(newDistance[0]);
    }
}

__global__ void CalculatePOS( int States, float *CentroidDist, float *POSOutput )
{
    extern __shared__ float shared[];
    float* distance = (float*)&shared;
    float* tPOS = (float*)&distance[States];
    int tid = threadIdx.x;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;

    while(tid < States)
    {
        distance[tid] = CentroidDist[bid*States+tid];
        tPOS[tid] = (float)(1.0/(1e-9+(double)distance[tid]));
        tid += blockDim.x;
    }
    __syncthreads();

    int dOld = States;
    int d = States >> 1;
    while (d != 0)
    {
        tid = threadIdx.x;
        dOld = dOld - d*2;
        while(tid < d)
        {
            tPOS[tid] += tPOS[tid + d];
            if (dOld == 1 && tid == d-1)
            {
                // special case of odd numbers
                tPOS[tid] += tPOS[tid + d + 1];
            }
            tid += blockDim.x;
        }
        // Sync moment before starting with next iteration of reduction.
        __syncthreads();

        dOld = d;
        d >>= 1;
    }

    tid = threadIdx.x;
    while(tid < States)
    {
        // This is the POS for all centroids (It looks like this is the input for the next layer also)
        // The output is missing the advice of higher layer
    	POSOutput[tid+bid*States] = (float)(1.0/(1e-9+(double)distance[tid]))/tPOS[0];
        tid += blockDim.x;
    }
}

/**
 * UpdateBeliefs - Performs the P(s'|s,a)*b(s) calculations of the DeSTIN belief update rule.
 *
 * PSSA means P(s'|s,a)*b(s) where a = advice, meaning
 * probability of transitioning to state s' given the current state s and the parents node's advice (or state) a.
 * Each node has a separate counting table for each possible parent advice state
 * of size N x N, where N is the number of centroids (states) of the child node.
 * The number of counting tables per node equal to the number of parent states.
 * Each time a node transitions from s to s' given advice a, the counting table for advice a
 * has the value of the element at row s and column s' incremented by 1. Then, to get the
 * probability, that value is divided by the corresponding value in the SumsTables.
 * There is one sum table per counting table, which has one element per column of the
 * matching counting table which sums up the elements of the column
 *
 * states - number of node centroids
 * dPOS - P(o|s') calculated from CalculatePOS kernel
 * dNewBeliefs - b'(s') - updated beliefs. The node output, fed to parent nodes as input
 * dOldBeliefs - b(s) - beliefs how they were before calling this kernel, currently dOldBeliefs points to same memory location as dNewBeliefs
 * dCountingTables - keeps track of the P(s'|s,a) table along with the dSumTables
 * dParentsInputAdvice - input advice from the parent node. The 'a' of P(s'|s,a). NULL if this is the top layer, no parent layer.
 * parentStates - number of centroids of the parent node. Expects it to be set to 1 if this is the top layer.
 * dSumTables - vector of the sum of the columns of the dCountingTables
 * isPSSATraining - if true, the PSSA counting tables used to calculate P(s'|s,a) are updated
 * dOutputAdvice - this node's advice to be fed to its children nodes
 */
//TODO: make a check to see if it has enough shared memory
__global__ void UpdateBeliefs( const int states, float *dPOS, float * dNewBeliefs, float * dOldBeliefs, int * dCountingTables,
		int * dParentsInputAdvice, int parentStates, int * dSumTables, bool isPSSATraining, int * dOutputAdvice){

	int bid = blockIdx.x + blockIdx.y * gridDim.x; //corresponds to the node

	//TODO: enforce square layers or update child to parent mapping code to handle non square layers
	//Points the 4 children nodes to the right parent for advice
	//Be careful of integer division if trying to simplify this.
	//TODO: simplify this
	int parent_node_id = blockIdx.x / 2 + (blockIdx.y /2 ) * (gridDim.x / 2) ;
	//TODO: could make one thread pull from global memory into shared memory, instead of all threads pulling
	const int advice = dParentsInputAdvice==NULL ? 0 : dParentsInputAdvice[parent_node_id];

	const int s2 = states * states;

	// Variable cts (counting table start) is the first element (at 0,0) of the correct PSSA counting table
	// based on the node and advice state.
	const int cts = bid * parentStates * s2 + advice * s2;


	extern __shared__ float cache[]; // the cache saves each P(s'|s,c)*b(s) for all s' and s for the given advice c. The size  is states x states ( plus another states tacked on the end later, see kernel launch params)

	//variable sp is read as "s prime" as in b'(s') which is the left side of the belief update equation.
	for(int sp = threadIdx.y; sp < states ; sp += blockDim.y) {
		int ctr = cts + sp * states; // ctr (counting table row) is the first element of the sp'th row of the a'th=advice counting table
		//s = current state
		for(int s = threadIdx.x ;  s < states ; s += blockDim.x ) {
			int i = ctr + s;//the s' row by s column element of the counting table
			//TODO: i might be performing this multiplication in the wrong order
			float prob = (float)dCountingTables[i] / (float) dSumTables[bid * parentStates * states + advice * states + s];
			//TODO: should probably save the dOldBeliefs vector to a shared memory variable first
			// to prevent having to pull it from global memory N=states times
			cache[ sp * states + s] = dOldBeliefs[bid * states + s] * prob; // this is the P(s'|s,c)*b(s) calculation.
		}
	}
	__syncthreads();



	//this part performs a reduction on the sums of the P(s'|s,c)*b(s) rows
	//of the cache table, storing the sums in the first column of the table.
	int dOld = states;
	for (int d = states >> 1;  d != 0; d >>= 1) { 				
		dOld -= d*2;	
		for(int sp = threadIdx.y; sp < states ; sp += blockDim.y){
			for(int s = threadIdx.x; s < d ; s += blockDim.x){
				int i = sp * states + s;
				cache[i] +=  cache[i + d];
				//trick for if cache has odd length
				if(dOld == 1 && s == d - 1){
					cache[i] += cache[i + d + 1];
				}
			}
		}
		__syncthreads(); 
		dOld = d;
	}

	float * pssc_b_vector = (float *)&cache[s2]; //length is states, start this vector right after the end of cache array

	//multiply the two parts of the belief update equation numerator together, Pr(o|s') by Sum[ Pr(s'|s,c)*b(S) ]
	//The cache[sp * states] is the Pr(s'|s,c)*b(S)  vector

	//we launched with a 2d block of threads now only dealing with 1d arrays, so convert this back to 1d so we waste fewer threads
	int sp_start  = threadIdx.y * blockDim.x + threadIdx.x; 
	int n_threads = blockDim.x * blockDim.y;

	//transform it from a column into a row
	for(int sp = sp_start ; sp < states ; sp += n_threads ){
		pssc_b_vector[sp] = cache[sp * states] *= dPOS[bid * states + sp];
	}
	__syncthreads(); //TODO: might not need this here


	//find the sum of the pssc_b_vector so it can be normalized   
	dOld = states;
	int s_start = sp_start;   
	for (int d = states >> 1; d!=0 ; dOld = d, d>>=1 ){
		dOld -= d*2;
		for(int s = s_start; s < d; s +=  n_threads ){
			pssc_b_vector[s] += pssc_b_vector[s + d ];
			if(dOld == 1 && s ==  d - 1){
				pssc_b_vector[s] += pssc_b_vector[s + d + 1];
			}
		}
		__syncthreads();	
	}

	float sum = pssc_b_vector[0];


	//pssc_b_vector will now contain b'(s') after normalizing the Pr(s'|s,c)*b(S) vector
	for(int sp = sp_start; sp < states ; sp += n_threads ){
		dNewBeliefs[ bid * states + sp] =  pssc_b_vector[sp] = cache[sp * states] /= sum;
	}

	//Update the p(s'|s,a) counting tables, incrementing the s column and s' row of the a'th counting table by one,
	//and incrementing the s column of the a'th sum tables by one
	int * max_index = (int *)cache; //max_index size = #states. Overwrite first row of cache shared memory to save winning index.
	//find max belief, store corresponding index in max_index[0]
	find_max(sp_start, n_threads, states, pssc_b_vector, max_index);
	//set max belief state as advice for child nodes
	if(sp_start == 0){//only one thread does this to save memory bandwidth
		int old_winning_belief = dOutputAdvice[bid];
		//new winning belief
		dOutputAdvice[bid] = max_index[0];
		if(isPSSATraining){
			updateCountingTables(states, parentStates, dCountingTables, advice, old_winning_belief, max_index[0], dSumTables, bid);
		}
	}

}

__device__ void updateCountingTables(int mStates, int parentStates, int * dCountingTables,
		int advice, int previousWinningBelief, int newWinningBelief, int * dSumTables, int bid){
	int s2 = mStates * mStates;
	//make sure Im consistent with old states across the top and new states down the side for the table.
	int i = bid * parentStates * s2 //node
		+ advice * s2 //advice table for node
		+ newWinningBelief * mStates //row of table
		+ previousWinningBelief;	//col of table

	dCountingTables[i]++;

	//dSumTables, collection of 1 dimensional vectors. Each node has the same number of them as its counting tables or one for each parent state.
	//One sum vector has length equal to the number of the node's centroids or states. Each element is the sum of the corresponding column
	//of the dCountingTable
	i = bid * parentStates * mStates // node index
		+ advice * mStates //advice index
		+ previousWinningBelief; //element of sum vector
	dSumTables[i]++;

}
/**
 * find_max - finds the maximum value of winner and its corresponding index. The 
 * maximum value is stored at winner[0] and the corresponding index of the maximum
 * value is stored at winnerId[0]
 * 
 * thread_start = tid
 * threadcount = how many threads
 * length = size of vector to find the maximum of
 * winner = input vector to find maximum of, winner[0] will contain the maximum value afterwords
 * winnerId = empty buffer to be used for scratch, winnerId[0] will have the max id afterwards
 */
__device__ void find_max(const int thread_start, const int threadcount, const int length,float * winner, int * winnerId ){

	int tid;
	for(tid = thread_start; tid < length ; tid += threadcount ){
		winnerId[tid] = tid;
	}
	__syncthreads();

	for(int dOld = length, d = length >> 1; d != 0 ; dOld = d, d >>= 1 ){
		for(tid = thread_start , dOld -= d*2 ; tid < d ; tid += threadcount){
			int tidd= tid + d;	
			if(winner[tid] < winner[tidd]){
				// Move large index to the beginning
				winner[tid] = winner[tidd];
				winnerId[tid] = winnerId[tidd];
			}
			if (dOld == 1 && tid == d-1){
				// special case of odd numbers
				if(winner[tid] < winner[tidd + 1]){
					winner[tid] = winner[tidd + 1];
					winnerId[tid] = winnerId[tidd + 1];
				}
			}
		}
		// Sync moment before starting with next iteration of reduction.
		__syncthreads();
	}
}

