#include "hip/hip_runtime.h"
#include "DestinData.h"

#include <fstream>
#include <map>
#include <math.h>
#include <iostream>

using namespace std;

DestinData::DestinData(void)
{
	mLastImageIndex=-1;
	mRows=0;
	mCols=0;
	// TODO: This is also not dynamic works for numbers of 32x32 images
    for(int r=0;r<40;r++)
    {
        for(int c=0;c<40;c++)
        {
            mImageWithOffset[r][c]=0;
        }
    }
	cout << "Destin data created" << endl;
}

DestinData::~DestinData(void)
{
	hipFree(dImage);
	delete [] mImage;
	cout << "Destin data deleted" << endl;
}

void DestinData::LoadFile(const char* sFileName)
{
	mImagePointer.clear();
	mLabels.clear();
	mUniqueLabels.clear();
	mMapLabelToIndexVector.clear();

	std::ifstream stmIn;
	stmIn.open(sFileName,ios::in | ios::binary );
	if(stmIn.is_open()){
		cout << "Found input file." << endl;
	}else{
		cout << "Did not find input file." << endl;
		return;
	}

	int iSignals;
	stmIn.read( (char*)&iSignals,sizeof(iSignals));
	stmIn.read( (char*)&mRows, sizeof(mRows) );
	stmIn.read( (char*)&mCols, sizeof(mCols) );
	// Create the array for pinned memory for CUDA
    int size = mRows*mCols;
    // Host side memory
    mImage = new float[size];
    // Device side memory
    hipMalloc( (void**)&dImage, size*sizeof(float) );

	int iLabel;
	unsigned char* cImageData;
	float** fImageDataByRow;
	float *fRow;

	map<int, vector<int> >::iterator it;

	for(int i=0; i<iSignals; i++)
	{
		stmIn.read( (char*)&iLabel, sizeof(iLabel) );
		mLabels.push_back(iLabel);
		it = mMapLabelToIndexVector.find(iLabel);
		if ( it==mMapLabelToIndexVector.end() ) //didn't find it
		{
			vector<int> vInt;
			vInt.push_back((int)(mLabels.size())-1);
			mMapLabelToIndexVector[iLabel]=vInt;
			mUniqueLabels.push_back(iLabel);
		}
		else
		{
			it->second.push_back((int)(mLabels.size())-1); //add the new vectors index...
		}
		cImageData = new unsigned char[mRows*mCols];
		stmIn.read( (char*)cImageData, mRows*mCols );  //The data is stored row 1, column 1-end, row 2, column 1-end, etc
		// so we have to transpose it here...
		fImageDataByRow = new float*[mRows];
		unsigned char* p = cImageData;
		for(int r=0;r<mRows;r++)
		{
			fRow = new float[mCols];
			for(int c=0;c<mCols;c++)
			{
				fRow[c]=((float)*p)/255.0;
				p++;
			}
			fImageDataByRow[r]=fRow;
		}
		delete cImageData;
		mImagePointer.push_back(fImageDataByRow);
	}
	stmIn.close();
	//hipMalloc();
	cout << "Finished reading file." << endl;
}

void DestinData::SetShiftedDeviceImage(int ImageIndex, int RowShift, int ColShift, int DemRow, int DemCol)
{
    // TODO: Might want to set C and R more dynamic. in case of different data set?
    int C = 4;
    int R = 4;
    // We don't have to load the image if it is the same one as before
    if ( ImageIndex!=mLastImageIndex )
    {
        // Load the image into the buffer with the "R,C" offset.
        float** fImage = mImagePointer[ImageIndex];
        for(int r=0;r<mRows;r++)
        {
            for(int c=0;c<mCols;c++)
            {
                mImageWithOffset[r+R][c+C]=fImage[r][c];
            }
        }
    }
    // Now load the data using the offset provided.
    // Convert a 2D array back to a 1D array
    int i = 0;
    for(int row=0;row<mRows;row+=DemRow)
    {
        for(int col=0;col<mCols;col+=DemCol)
        {
            // To optimize the memory use inside CUDA put the DemRow*DemCol as one block.
            // This makes the kernel do the same for all layers also.
            for(int r=0;r<DemRow;r++)
            {
                for(int c=0;c<DemCol;c++)
                {
                    mImage[i]=mImageWithOffset[r+row+RowShift][c+col+ColShift];
                    i++;
                }
            }
        }
    }
    // Copy data from host to device
    hipMemcpy( dImage, mImage, mRows*mCols*sizeof(float), hipMemcpyHostToDevice );
    mLastImageIndex=ImageIndex;
}

void DestinData::GetLabelList(vector<int>& Labels)
{
    Labels.clear();
    for(int i=0;i<(int)(mLabels.size());i++)
    {
        Labels.push_back(mLabels[i]);
    }
}

void DestinData::GetUniqueLabels(vector<int>& vUniqueLabels)
{
    vUniqueLabels.clear();
	vector<int>::iterator it = this->mUniqueLabels.begin();
	while ( it != mUniqueLabels.end() )
	{
	    vUniqueLabels.push_back(*it++);
	}

}

void DestinData::GetIndicesForThisLabel(int iLabel, vector<int>& IndicesForThisLabel )
{
	IndicesForThisLabel.clear();

	map<int, vector<int> >::iterator it;

	it = mMapLabelToIndexVector.find(iLabel);
	if ( it!=mMapLabelToIndexVector.end() ) //find it
	{
		vector<int>::iterator vit;
		vit = it->second.begin();
		while ( vit != it->second.end() )
		{
			IndicesForThisLabel.push_back( *vit++ );
		}
	}
}

void DestinData::DoSpecial4x4FFT(float** &fSubImage, float* fUniqueVector)
{
	// I don't generally recommend doing FFTs like this, but since we only want 4x4 I didn't want to 
	// go to the trouble of adding an external library.  Plus you can do 4x4 with only adds & subtracts...
	int r,c;
	float fV;
	//Initialize the imaginary parts that we won't be 'hitting'...
	mIP[0][0]=0;mIP[2][0]=0;mIP[0][2]=0;mIP[2][2]=0;
	////////////////Row 0 Col 0///////
	fV = fSubImage[0][0];
	mRP[0][0]=fV; mRP[0][0]=fV; mRP[1][0]=fV; mRP[1][0]=fV; mRP[2][0]=fV; mRP[2][0]=fV; mRP[3][0]=fV; mRP[3][0]=fV; 
	mRP[0][1]=fV; mRP[0][1]=fV; mRP[1][1]=fV; mRP[1][1]=fV; mRP[2][1]=fV; mRP[2][1]=fV; mRP[3][1]=fV; mRP[3][1]=fV; 
	mRP[0][2]=fV; mRP[0][2]=fV; mRP[1][2]=fV; mRP[1][2]=fV; mRP[2][2]=fV; mRP[2][2]=fV; mRP[3][2]=fV; mRP[3][2]=fV; 
	mRP[0][3]=fV; mRP[0][3]=fV; mRP[1][3]=fV; mRP[1][3]=fV; mRP[2][3]=fV; mRP[2][3]=fV; mRP[3][3]=fV; mRP[3][3]=fV; 
	////////////////Row 0 Col 1///////
	fV = fSubImage[0][1];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]+fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]+fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[1][2]=mRP[1][2]-fV; mRP[2][2]=mRP[2][2]-fV; mRP[3][2]=mRP[3][2]-fV; 
	mIP[0][1]=-fV; mIP[0][1]=-fV; mIP[1][1]=-fV; mIP[1][1]=-fV; mIP[2][1]=-fV; mIP[2][1]=-fV; mIP[3][1]=-fV; mIP[3][1]=-fV; 
	mIP[0][3]=fV; mIP[0][3]=fV; mIP[1][3]=fV; mIP[1][3]=fV; mIP[2][3]=fV; mIP[2][3]=fV; mIP[3][3]=fV; mIP[3][3]=fV; 
	////////////////Row 0 Col 2///////
	fV = fSubImage[0][2];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]+fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]+fV; 
	mRP[0][1]=mRP[0][1]-fV; mRP[1][1]=mRP[1][1]-fV; mRP[2][1]=mRP[2][1]-fV; mRP[3][1]=mRP[3][1]-fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[1][2]=mRP[1][2]+fV; mRP[2][2]=mRP[2][2]+fV; mRP[3][2]=mRP[3][2]+fV; 
	mRP[0][3]=mRP[0][3]-fV; mRP[1][3]=mRP[1][3]-fV; mRP[2][3]=mRP[2][3]-fV; mRP[3][3]=mRP[3][3]-fV; 
	////////////////Row 0 Col 3///////
	fV = fSubImage[0][3];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]+fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]+fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[1][2]=mRP[1][2]-fV; mRP[2][2]=mRP[2][2]-fV; mRP[3][2]=mRP[3][2]-fV; 
	mIP[0][1]=mIP[0][1]+fV; mIP[1][1]=mIP[1][1]+fV; mIP[2][1]=mIP[2][1]+fV; mIP[3][1]=mIP[3][1]+fV; 
	mIP[0][3]=mIP[0][3]-fV; mIP[1][3]=mIP[1][3]-fV; mIP[2][3]=mIP[2][3]-fV; mIP[3][3]=mIP[3][3]-fV; 
	////////////////Row 1 Col 0///////
	fV = fSubImage[1][0];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[0][1]=mRP[0][1]+fV; mRP[2][1]=mRP[2][1]-fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[2][2]=mRP[2][2]-fV; mRP[0][3]=mRP[0][3]+fV; mRP[2][3]=mRP[2][3]-fV; 
	mIP[1][0]=-fV; mIP[1][0]=-fV; mIP[3][0]=fV; mIP[3][0]=fV; mIP[1][1]=mIP[1][1]-fV; mIP[3][1]=mIP[3][1]+fV; 
	mIP[1][2]=-fV; mIP[1][2]=-fV; mIP[3][2]=fV; mIP[3][2]=fV; mIP[1][3]=mIP[1][3]-fV; mIP[3][3]=mIP[3][3]+fV; 
	////////////////Row 1 Col 1///////
	fV = fSubImage[1][1];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[1][1]=mRP[1][1]-fV; mRP[3][1]=mRP[3][1]+fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[2][2]=mRP[2][2]+fV; mRP[1][3]=mRP[1][3]+fV; mRP[3][3]=mRP[3][3]-fV; 
	mIP[1][0]=mIP[1][0]-fV; mIP[3][0]=mIP[3][0]+fV; mIP[0][1]=mIP[0][1]-fV; mIP[2][1]=mIP[2][1]+fV; 
	mIP[1][2]=mIP[1][2]+fV; mIP[3][2]=mIP[3][2]-fV; mIP[0][3]=mIP[0][3]+fV; mIP[2][3]=mIP[2][3]-fV; 
	////////////////Row 1 Col 2///////
	fV = fSubImage[1][2];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[0][1]=mRP[0][1]-fV; mRP[2][1]=mRP[2][1]+fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[2][2]=mRP[2][2]-fV; mRP[0][3]=mRP[0][3]-fV; mRP[2][3]=mRP[2][3]+fV; 
	mIP[1][0]=mIP[1][0]-fV; mIP[3][0]=mIP[3][0]+fV; mIP[1][1]=mIP[1][1]+fV; mIP[3][1]=mIP[3][1]-fV; 
	mIP[1][2]=mIP[1][2]-fV; mIP[3][2]=mIP[3][2]+fV; mIP[1][3]=mIP[1][3]+fV; mIP[3][3]=mIP[3][3]-fV; 
	////////////////Row 1 Col 3///////
	fV = fSubImage[1][3];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[1][1]=mRP[1][1]+fV; mRP[3][1]=mRP[3][1]-fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[2][2]=mRP[2][2]+fV; mRP[1][3]=mRP[1][3]-fV; mRP[3][3]=mRP[3][3]+fV; 
	mIP[1][0]=mIP[1][0]-fV; mIP[3][0]=mIP[3][0]+fV; mIP[0][1]=mIP[0][1]+fV; mIP[2][1]=mIP[2][1]-fV; 
	mIP[1][2]=mIP[1][2]+fV; mIP[3][2]=mIP[3][2]-fV; mIP[0][3]=mIP[0][3]-fV; mIP[2][3]=mIP[2][3]+fV; 
	////////////////Row 2 Col 0///////
	fV = fSubImage[2][0];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]-fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]-fV; 
	mRP[0][1]=mRP[0][1]+fV; mRP[1][1]=mRP[1][1]-fV; mRP[2][1]=mRP[2][1]+fV; mRP[3][1]=mRP[3][1]-fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[1][2]=mRP[1][2]-fV; mRP[2][2]=mRP[2][2]+fV; mRP[3][2]=mRP[3][2]-fV; 
	mRP[0][3]=mRP[0][3]+fV; mRP[1][3]=mRP[1][3]-fV; mRP[2][3]=mRP[2][3]+fV; mRP[3][3]=mRP[3][3]-fV; 
	////////////////Row 2 Col 1///////
	fV = fSubImage[2][1];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]-fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]-fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[1][2]=mRP[1][2]+fV; mRP[2][2]=mRP[2][2]-fV; mRP[3][2]=mRP[3][2]+fV; 
	mIP[0][1]=mIP[0][1]-fV; mIP[1][1]=mIP[1][1]+fV; mIP[2][1]=mIP[2][1]-fV; mIP[3][1]=mIP[3][1]+fV; 
	mIP[0][3]=mIP[0][3]+fV; mIP[1][3]=mIP[1][3]-fV; mIP[2][3]=mIP[2][3]+fV; mIP[3][3]=mIP[3][3]-fV; 
	////////////////Row 2 Col 2///////
	fV = fSubImage[2][2];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]-fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]-fV; 
	mRP[0][1]=mRP[0][1]-fV; mRP[1][1]=mRP[1][1]+fV; mRP[2][1]=mRP[2][1]-fV; mRP[3][1]=mRP[3][1]+fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[1][2]=mRP[1][2]-fV; mRP[2][2]=mRP[2][2]+fV; mRP[3][2]=mRP[3][2]-fV; 
	mRP[0][3]=mRP[0][3]-fV; mRP[1][3]=mRP[1][3]+fV; mRP[2][3]=mRP[2][3]-fV; mRP[3][3]=mRP[3][3]+fV; 
	////////////////Row 2 Col 3///////
	fV = fSubImage[2][3];
	mRP[0][0]=mRP[0][0]+fV; mRP[1][0]=mRP[1][0]-fV; mRP[2][0]=mRP[2][0]+fV; mRP[3][0]=mRP[3][0]-fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[1][2]=mRP[1][2]+fV; mRP[2][2]=mRP[2][2]-fV; mRP[3][2]=mRP[3][2]+fV; 
	mIP[0][1]=mIP[0][1]+fV; mIP[1][1]=mIP[1][1]-fV; mIP[2][1]=mIP[2][1]+fV; mIP[3][1]=mIP[3][1]-fV; 
	mIP[0][3]=mIP[0][3]-fV; mIP[1][3]=mIP[1][3]+fV; mIP[2][3]=mIP[2][3]-fV; mIP[3][3]=mIP[3][3]+fV; 
	////////////////Row 3 Col 0///////
	fV = fSubImage[3][0];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[0][1]=mRP[0][1]+fV; mRP[2][1]=mRP[2][1]-fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[2][2]=mRP[2][2]-fV; mRP[0][3]=mRP[0][3]+fV; mRP[2][3]=mRP[2][3]-fV; 
	mIP[1][0]=mIP[1][0]+fV; mIP[3][0]=mIP[3][0]-fV; mIP[1][1]=mIP[1][1]+fV; mIP[3][1]=mIP[3][1]-fV; 
	mIP[1][2]=mIP[1][2]+fV; mIP[3][2]=mIP[3][2]-fV; mIP[1][3]=mIP[1][3]+fV; mIP[3][3]=mIP[3][3]-fV; 
	////////////////Row 3 Col 1///////
	fV = fSubImage[3][1];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[1][1]=mRP[1][1]+fV; mRP[3][1]=mRP[3][1]-fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[2][2]=mRP[2][2]+fV; mRP[1][3]=mRP[1][3]-fV; mRP[3][3]=mRP[3][3]+fV; 
	mIP[1][0]=mIP[1][0]+fV; mIP[3][0]=mIP[3][0]-fV; mIP[0][1]=mIP[0][1]-fV; mIP[2][1]=mIP[2][1]+fV; 
	mIP[1][2]=mIP[1][2]-fV; mIP[3][2]=mIP[3][2]+fV; mIP[0][3]=mIP[0][3]+fV; mIP[2][3]=mIP[2][3]-fV; 
	////////////////Row 3 Col 2///////
	fV = fSubImage[3][2];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[0][1]=mRP[0][1]-fV; mRP[2][1]=mRP[2][1]+fV; 
	mRP[0][2]=mRP[0][2]+fV; mRP[2][2]=mRP[2][2]-fV; mRP[0][3]=mRP[0][3]-fV; mRP[2][3]=mRP[2][3]+fV; 
	mIP[1][0]=mIP[1][0]+fV; mIP[3][0]=mIP[3][0]-fV; mIP[1][1]=mIP[1][1]-fV; mIP[3][1]=mIP[3][1]+fV; 
	mIP[1][2]=mIP[1][2]+fV; mIP[3][2]=mIP[3][2]-fV; mIP[1][3]=mIP[1][3]-fV; mIP[3][3]=mIP[3][3]+fV; 
	////////////////Row 3 Col 3///////
	fV = fSubImage[3][3];
	mRP[0][0]=mRP[0][0]+fV; mRP[2][0]=mRP[2][0]-fV; mRP[1][1]=mRP[1][1]-fV; mRP[3][1]=mRP[3][1]+fV; 
	mRP[0][2]=mRP[0][2]-fV; mRP[2][2]=mRP[2][2]+fV; mRP[1][3]=mRP[1][3]+fV; mRP[3][3]=mRP[3][3]-fV; 
	mIP[1][0]=mIP[1][0]+fV; mIP[3][0]=mIP[3][0]-fV; mIP[0][1]=mIP[0][1]+fV; mIP[2][1]=mIP[2][1]-fV; 
	mIP[1][2]=mIP[1][2]-fV; mIP[3][2]=mIP[3][2]+fV; mIP[0][3]=mIP[0][3]-fV; mIP[2][3]=mIP[2][3]+fV;

	//Finally take the magnitude...
	for(r=0;r<4;r++)
	{
		for(c=0;c<4;c++)
		{
			fSubImage[r][c]=(float)pow((double)(mRP[r][c]*mRP[r][c]+mIP[r][c]*mIP[r][c]),(double)0.5);
		}
	}

	//Unique values are 10: (0,0)-(2,2) and also (3,1)
	float* p = fUniqueVector;
	*p=fSubImage[0][0]/10.0; p++;
	*p=fSubImage[1][0]/10.0; p++;
	*p=fSubImage[2][0]/10.0; p++;

	*p=fSubImage[0][1]/10.0; p++;
	*p=fSubImage[1][1]/10.0; p++;
	*p=fSubImage[2][1]/10.0; p++;

	*p=fSubImage[0][2]/10.0; p++;
	*p=fSubImage[1][2]/10.0; p++;
	*p=fSubImage[2][2]/10.0; p++;

	*p=fSubImage[3][1]/10.0;
}
