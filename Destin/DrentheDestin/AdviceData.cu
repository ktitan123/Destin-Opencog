#include "AdviceData.h"

#include <math.h>
#include <iostream>
#include <stdio.h>
#include <vector>
#include <fstream>
#include <ctime>
#include <sys/stat.h>

#ifdef _WIN32
#include <direct.h>
#include <string>
#else
// Linux only requirements...
#include <string.h>
#include <stdlib.h>
#endif

using namespace std;

AdviceData::AdviceData(void)
{
	mParentBeliefs = NULL;
	mParentUnsupervisedMemberships = NULL;
	ClearAdvice();
	mTempID = -1;
	mCountNumber = -1;
}


void AdviceData::SetID(int L,int R,int C)
{ 
	mTempID = 1000*L+100*R+C; 
};


void AdviceData::SetMovementAndBestPSSATableIndex(int Movement,int Index)
{
	mMovementBeingProcessed=Movement; 
	if ( mMovementBeingProcessed==0 )
	{
		mCountNumber++; 
	}

	mWinningLabel = Index;
}


AdviceData::~AdviceData(void)
{
}

AdviceData::AdviceData(AdviceData& oN)
{
	mParentBeliefs = NULL;
	mParentUnsupervisedMemberships = NULL;
	ClearAdvice();
	*this = oN;
}

AdviceData& AdviceData::operator = (AdviceData& oN)
{
	mTempID = oN.mTempID;
	mCountNumber = oN.mCountNumber;

	ClearAdvice();
	mWinningLabel = oN.mWinningLabel;
	mMovementBeingProcessed = oN.mMovementBeingProcessed;
	mbValidParentUnsupervisedAdvice = oN.mbValidParentUnsupervisedAdvice;

	mNumberOfParentUnsupervisedMemberships = oN.mNumberOfParentUnsupervisedMemberships;
	mNumberOfParentBeliefs = oN.mNumberOfParentBeliefs;

	if ( mNumberOfParentBeliefs != 0 )
	{
		mParentBeliefs = new float[mNumberOfParentBeliefs];
		memcpy( mParentBeliefs,oN.mParentBeliefs,sizeof(*mParentBeliefs)*mNumberOfParentBeliefs);
	}

	if ( mNumberOfParentUnsupervisedMemberships != 0 )
	{
	   mParentUnsupervisedMemberships = new float[mNumberOfParentUnsupervisedMemberships];
		memcpy( mParentUnsupervisedMemberships,oN.mParentUnsupervisedMemberships,sizeof(*mParentUnsupervisedMemberships)*mNumberOfParentUnsupervisedMemberships);
	}
	return *this;
}

bool AdviceData::operator == (AdviceData& o)
{
	if ( mWinningLabel!=o.mWinningLabel ) return false;
	if ( mMovementBeingProcessed != o.mMovementBeingProcessed ) return false;
	return true;
}

bool AdviceData::operator != (AdviceData& o)
{
	return !(*this==o);
}

void AdviceData::ClearAdvice()
{
	mbValidParentUnsupervisedAdvice = false;
	mWinningLabel=0;
	mMovementBeingProcessed = -1;
	mNumberOfParentUnsupervisedMemberships=0;
	mNumberOfParentBeliefs=0;
	if ( mParentBeliefs != NULL )
	{
		delete mParentBeliefs;
		mParentBeliefs=NULL;
	}
	if ( mParentUnsupervisedMemberships != NULL )
	{
		delete mParentUnsupervisedMemberships;
		mParentUnsupervisedMemberships=NULL;
	}

}

bool AdviceData::GetValidParentUnsupervisedAdvice()
{
	return mbValidParentUnsupervisedAdvice;
}

bool AdviceData::WriteToStream(std::ofstream& stmOutput)
{
	int iFlag = -2020;
	stmOutput.write( (char*)&iFlag, sizeof(iFlag) );
	stmOutput.write( (char*)&mWinningLabel, sizeof(mWinningLabel) );
	stmOutput.write( (char*)&mMovementBeingProcessed, sizeof(mMovementBeingProcessed) );
	return true;
}

void AdviceData::SetParentBeliefs( float* pBeliefs, int iN )
{
	if ( mParentBeliefs==NULL )
	{
		mParentBeliefs = new float[iN];
	}
	memcpy( mParentBeliefs, pBeliefs, iN*sizeof(*pBeliefs) );
	mNumberOfParentBeliefs=iN;
}

void AdviceData::SetParentUnsupervisedAdvice( float* pUnsup, int iN )
{
	if ( pUnsup==NULL )
	{
		mbValidParentUnsupervisedAdvice = false;
	}
	else
	{
		if ( mParentUnsupervisedMemberships==NULL )
		{
			mParentUnsupervisedMemberships = new float[iN];
		}
		memcpy( mParentUnsupervisedMemberships, pUnsup, iN*sizeof(*pUnsup) );
		mbValidParentUnsupervisedAdvice = true;
		mNumberOfParentUnsupervisedMemberships = iN;
	}
}

bool AdviceData::ReadFromStream(std::ifstream& stmInput)
{
	if ( mParentBeliefs != NULL )
	{
		delete mParentBeliefs;
		mParentBeliefs=NULL;
	}
	if ( mParentUnsupervisedMemberships != NULL )
	{
		mParentUnsupervisedMemberships=NULL;
		delete mParentUnsupervisedMemberships;
	}
	ClearAdvice();
	int iFlag;
	stmInput.read( (char*)&iFlag, sizeof(iFlag) );
	if ( iFlag==-2020 )
	{
		stmInput.read( (char*)&mWinningLabel, sizeof(mWinningLabel) );
		stmInput.read( (char*)&mMovementBeingProcessed, sizeof(mMovementBeingProcessed) );
	}
	else
	{
		throw "Exception reading AdviceData from stream!";
	}
	return true;
}
