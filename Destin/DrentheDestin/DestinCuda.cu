#include "hip/hip_runtime.h"
#include "DestinCuda.h"


using namespace std;

// TODO: This main file is still pretty messy.
/*
 * It's almost a copy of the original main of DeSTIN where the custom file reader is replaced by a XML reader(pugixml)
 * Because of the quite basic understanding of the code and a more overall understanding of DeSTIN there are a lot of variables not been used.
 * Also cause there is still work in progress.
 * It still might be a idea to just replace the command line complete and do everything from the configuration
 * This will save a lot of code to analyze the input arguments
 */
void DestinCuda::PrintHelp()
{
    // ***************************
    // Print out how to use DeSTIN
    // ***************************

    cout << "Usage: DestinCuda CodeWord MAXCNT LayerToShow ParamsFile TrainingDataFile DestinOutputFile TargetDirectory [OutputDistillationLevel]" << endl;
    cout << "Where:" << endl;
    cout << "    CodeWord must have 11 digits RRRRXXYYYYY" << endl;
    cout << "        RRRR  = 0000 to 9999 where 0000 is real random time." << endl;
    cout << "        XX    = 01 to 99 number of classes will not be higher then training file." << endl;
    cout << "        YYYYY = 00000 to 99999 number of examples of each class." << endl;
    cout << "                00000 means RANDOMLY PICK EXAMPLES until we finish clustering, period, up to max iterations." << endl;
    cout << "    MAXCNT is the number of digits we show it to train the unsupervised DeSTIN architecture" << endl;
    cout << "    LayerToShow = layer written to output file; it is given as S:E:O:P:T where " << endl;
    cout << "        S = first layer to write" << endl;
    cout << "        E = last layer to write" << endl;
    cout << "        O = offset for movements to write" << endl;
    cout << "        P = period of movements to write" << endl;
    cout << "        T = type.  Nothing (and no !) is beliefs.  Type can be: " << endl;
    cout << "            A is belief in advice states computed by tabular method." << endl;
    cout << "            N is belief in advice states computed by neural network function approximator." << endl;
    cout << "            L is belief in advice states computed by linear function approximator." << endl;
    cout << "    ParamsFile is a file that has the run parameters" << endl;
    cout << "    TrainingDataFile is the binary data file for training.  A testing file with the SAME NAME and appended with _TESTING is assumed" << endl;
    cout << "    DestinOutputFile is the name of the DeSTIN network output file for saving." << endl;
    cout << "         Use -D as default, which is the experiment number with a .xml at the end, in the TargetDirectory directory" << endl;
    cout << "    TargetDirectory is where we want to put the MAIN OUTPUT DATA FILES.  We ALWAYS write an experiment marker to the " << endl;
    cout << "        ../DiagnosticData area.  But if you are writing out a lot of data you can specify another directory." << endl;
    cout << "        Put D for default which is the ../DiagnosticData area." << endl;
    cout << "    [OutputDistillationLevel] is optional.  If this exists it must be a number and currently its got to be 0.  "<<endl;
    cout << "        0 = regular outputs with a lot of details about movements and processing: this is our input to SampleAndStack"<<endl;
    cout << "        1 = outputs compatible with the regular distilled output of SampleAndStack. If you use this you can skip SampleAndStack.exe" << endl;
}

bool DestinCuda::FileExists(string strFilename)
{
    // **************************
    // Does the given file exists
    // **************************
    // For detailed information look the return values of stat

    struct stat stFileInfo;
    bool blnReturn;
    int intStat;

    // Attempt to get the file attributes
    intStat = stat(strFilename.c_str(),&stFileInfo);
    if(intStat == 0) {
        // File exists
        blnReturn = true;
    }
    else
    {
        // File not exists or no permission
        blnReturn = false;
    }

    return(blnReturn);
}

string DestinCuda::GetNextFileForDiagnostic()
{
    // *************************************
    // Find next available experimental file
    // *************************************
    // Check if there is a previous experiment inside ../DiagnosticData

    string strFileName;
    int iExperimentNumber=-1;
    bool bFileFound = true;
    while ( bFileFound )
    {
        iExperimentNumber++;
        stringstream buffer;
        buffer << "../DiagnosticData/DestinDiagnostics" << iExperimentNumber << "-0.xml";
        strFileName =  buffer.str();

        bFileFound = FileExists(strFileName);
        strFileName.erase(strFileName.length()-6,2);
    }
    strFileName = strFileName.substr(18);

    return strFileName;
}

void DestinCuda::GetParameters( const char* cFilename, int& NumberOfLayers, double*& dcMu, double*& dcSigma, double*& dcRho,
                    int*& NumberOfStates, bool& bAveraging,bool& bFFT,bool& bBinaryPOS,int* DistanceMeasureArray,
                    bool& bUseStarvationTrace,int& PSSAUpdateDelay,bool& bIgnoreAdvice,
                    int**& SEQ, int& SEQ_LENGTH, string& sFileContents, int& iBlocksToProcess,
                    bool& bBasicOnlineClustering,
                    bool& bClanDestin, bool& bInitialLayerIsTransformOnly,bool& bUseGoodPOSMethod,
                    int*& RowsPerLayer, float*& FixedLearningRateLayer, bool*& bSelfAndUpperFeedback, int& LastLayerInputX, int& LastLayerInputY )
{
    // ******************************************
    // Read the XML config file (parameters file)
    // ******************************************
    // This function is rewritten and is not backwards compatible with the DestinPort one.
    // Instead of a txt file its now a XML file pugixml is used for parsing it.
    ifstream stmInput(cFilename);
    string sBuffer;
    // Put the config file into a vector and as one big string back to sFileCOntents
    while ( getline(stmInput, sBuffer) )
    {
        sFileContents = sFileContents + "~" + sBuffer + "\n";
    }
    stmInput.close();

    pugi::xml_document xFile;

    pugi::xml_parse_result result = xFile.load_file(cFilename);
    std::cout << "XML config file Load result: " << result.description() << endl;
    if ( result )
    {
        // Root node is destin
        pugi::xml_node root = xFile.child("destin");

        // Retrieve SEQ_LENGTH
        pugi::xml_node seq = root.child("seq");
        SEQ_LENGTH = seq.attribute("length").as_int();

        // Retrieve all steps
        SEQ = new int*[SEQ_LENGTH];
        pugi::xml_node step = seq.child("step");
        for( int iStep = 0; iStep < SEQ_LENGTH; iStep++ )
        {
            SEQ[iStep]=new int[2];
            SEQ[iStep][0] = step.attribute("x").as_int();
            SEQ[iStep][1] = step.attribute("y").as_int();
            step = step.next_sibling("step");
        }

        // Retrieve amount of layers
        pugi::xml_node layers = root.child("layers");
        NumberOfLayers = layers.attribute("value").as_int();
        LastLayerInputX = layers.attribute("inputX").as_int();
        LastLayerInputY = layers.attribute("inputY").as_int();

        // Retrieve configuration each layer
        dcMu = new double[NumberOfLayers];
        dcSigma = new double[NumberOfLayers];
        dcRho = new double[NumberOfLayers];
        NumberOfStates = new int[NumberOfLayers];
        DistanceMeasureArray = new int[NumberOfLayers];
        RowsPerLayer = new int[NumberOfLayers];
        FixedLearningRateLayer = new float[NumberOfLayers];
        bSelfAndUpperFeedback = new bool[NumberOfLayers];
        pugi::xml_node layer = layers.child("layer");
        // Loop true each layer configuration
        for( int iLayer = 0; iLayer < NumberOfLayers; iLayer++ )
        {
            dcMu[iLayer] = layer.attribute("mu").as_double();
            dcSigma[iLayer] = layer.attribute("sigma").as_double();
            dcRho[iLayer] = layer.attribute("rho").as_double();
            NumberOfStates[iLayer] = layer.attribute("states").as_int();
            DistanceMeasureArray[iLayer] = layer.attribute("distance").as_int();
            RowsPerLayer[iLayer] = layer.attribute("rowsColmsPerLayer").as_int();
            FixedLearningRateLayer[iLayer] = layer.attribute("fixedLearningRate").as_float();
            bSelfAndUpperFeedback[iLayer] = layer.attribute("selfAndUpperFeedback").as_bool();
            layer = layer.next_sibling("layer");
        }

        // Retrieve settings for overal DeSTIN
        pugi::xml_node settings = root.child("settings");
        bAveraging = settings.child("averaging").attribute("value").as_bool();
        bFFT = settings.child("fft").attribute("value").as_bool();
        bBinaryPOS = settings.child("binaryPos").attribute("value").as_bool();
        bUseStarvationTrace = settings.child("starvationTrace").attribute("value").as_bool();
        PSSAUpdateDelay = settings.child("pssaDelay").attribute("value").as_int();
        bIgnoreAdvice = settings.child("ignoreAdvice").attribute("value").as_bool();
        iBlocksToProcess = settings.child("processingBlockSize").attribute("value").as_int();
        bBasicOnlineClustering = settings.child("basicOnlineClustering").attribute("value").as_bool();
        bClanDestin = settings.child("clanDestin").attribute("value").as_bool();
        bInitialLayerIsTransformOnly = settings.child("initialLayerIsTransformOnly").attribute("value").as_bool();
        bUseGoodPOSMethod = settings.child("useGoodPOSMethod").attribute("value").as_bool();
    }
    else
    {
        std::cout << "Error description: " << result.description() << "\n";
        std::cout << "Error offset: " << result.offset << " (error at [..." << (cFilename + result.offset) << "]\n\n";
    }
    cout << "------------------" << endl;
}

bool DestinCuda::CreateDestinOnTheFly(string ParametersFileName, int& NumberOfLayers, DestinKernel*& DKernel,
                          DestinData& DataSourceForTraining, int& SEQ_LENGTH, int**& SEQ,
                          int*& ImageInput)

{
    // *********************
    // Create DeSTIN network
    // *********************

    double* dcMu;
    double* dcSigma;
    double* dcRho;
    int* NumberOfCentroids;
    bool bAveraging;
    bool bFFT;
    bool bBinaryPOS;
    int DistanceMeasureArray[128];
    bool bUseStarvationTrace;
    int PSSAUpdateDelay;
    bool bIgnoreAdvice;
    string sParametersFileContents;
    int iBlocksToProcess;
    bool bBasicOnlineClustering;
    bool bClanDestin;
    bool bInitialLayerIsTransformOnly;
    bool bDoGoodPOS;
    int* RowsPerLayer;
    float* FixedLearningRateLayer;
    bool* bSelfAndUpperFeedback;
    ImageInput = new int[2];

    GetParameters( ParametersFileName.c_str(), NumberOfLayers, dcMu, dcSigma, dcRho, NumberOfCentroids,
                   bAveraging, bFFT, bBinaryPOS, DistanceMeasureArray,
                   bUseStarvationTrace, PSSAUpdateDelay, bIgnoreAdvice, SEQ, SEQ_LENGTH,
                   sParametersFileContents, iBlocksToProcess,
                   bBasicOnlineClustering, bClanDestin, bInitialLayerIsTransformOnly, bDoGoodPOS,
                   RowsPerLayer, FixedLearningRateLayer, bSelfAndUpperFeedback, ImageInput[0], ImageInput[1]);


    DKernel = new DestinKernel[NumberOfLayers];
    //TODO: I think all these "new" declarations are not being deleted and potentially leaking memory
    int* ColsPerLayer = new int[NumberOfLayers];
    int* InputDimensionality = new int[NumberOfLayers];

    InputDimensionality[0] = 16; //4x4 has 16 inputs

    for(int Layer=1; Layer<NumberOfLayers; Layer++ ){
        InputDimensionality[Layer] = 4*NumberOfCentroids[Layer-1];
    }

    // hiprandGenerator_t is a CUDA version of rand
    // This fills the whole memory block with number between 0.0 and 1.0
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    // TODO: Add seed code instead of 1
    // This is the right place to do this saves the most time creating numbers. (Inside layer increase the time by +/- 5 times)
    hiprandSetPseudoRandomGeneratorSeed( gen, 1 );

    for( int Layer = NumberOfLayers - 1; Layer>=0 ; Layer-- ){
        ColsPerLayer[Layer] = RowsPerLayer[Layer];
        int parentStates = Layer == NumberOfLayers - 1 ? 1 : NumberOfCentroids[Layer + 1];
        DKernel[Layer].Create(
        		Layer, RowsPerLayer[Layer], ColsPerLayer[Layer],
        		NumberOfCentroids[Layer],parentStates,  InputDimensionality[Layer],
        		FixedLearningRateLayer[Layer], gen);

        //give the child layers their parents' advice
        if(Layer == NumberOfLayers - 1){
        	DKernel[Layer].SetInputAdvice(NULL);
        }else{
            DKernel[Layer].SetInputAdvice(DKernel[Layer+1].GetOutputAdvice());
        }
    }

    // The generator have to be destroyed after use.
    hiprandDestroyGenerator( gen );
    cout << "------------------" << endl;
    return 0;
}
/**
 *  parseCommandArgs
 *
 *  Parses the program command line arguments and fills
 *  the out CommandArgsStuc appropriately
 *      
 */

CommandArgsStuc createCommandArgsStuc(string sCodeWord, int MAX_CNT, string ParametersFileName){
    CommandArgsStuc cas;
    cas.sCodeWord = sCodeWord;
    
    return cas;
}

int DestinCuda::parseCommandArgs(int argc, char* argv[], CommandArgsStuc &out) {

 
    out.sCodeWord = argv[1];
    if (out.sCodeWord.length() != 11) {
        PrintHelp();
        return 1;
    }
    out.seed = atoi(argv[1]);
    out.MAX_CNT = atoi(argv[2]);
    
    
    // Argument: ParamsFile
    // A configuration file for DeSTIN
    out.ParametersFileName = argv[4];
    if (!FileExists(out.ParametersFileName)) {
        // According to the help the ParamsFile is always used? Maybe some vital information on how to load data?
        // Or some testing to see how the network reacts when expanding or shrinking the network.
        cout << "Parameters file name does not exist" << endl;
        return 1;
    }

     out.strDestinTrainingFileName = argv[5];
     
     
    // Argument: TargetDirectory
    // A given location instead or default
    string strDiagnosticDirectoryForData;
    string strArg7 = argv[7];
    if (strArg7 == "D") {
        strDiagnosticDirectoryForData = "../DiagnosticData/";
    } else {
        // Buffer with path + filename where to put diagnostic data
        stringstream buffer;
        buffer << strArg7.c_str() << "/";
        strDiagnosticDirectoryForData = buffer.str();
    }
    
   

    // Argument: DestinOutputFile
    string strDestinNetworkFileToWrite = argv[6]; // we write to this file, and then we read from it too!!
    if (strDestinNetworkFileToWrite == "-D") {
        // If given -D
        strDestinNetworkFileToWrite = strDiagnosticDirectoryForData + GetNextFileForDiagnostic();
        cout << "Writing default destin file to: " << strDestinNetworkFileToWrite << endl;
    }
    out.strDestinNetworkFileToRead = strDestinNetworkFileToWrite;

    

    out.strTesting = out.strDestinTrainingFileName + "_TESTING";
    return 0;
}


int DestinCuda::MainDestinExperiments(CommandArgsStuc & argsStruc)
{
    time_t destinStart = time(NULL);
    // ********************************************
    // Main experiment of DeSTIN (Also called main)
    // ********************************************


    // **********************
    // Loading data source(s)
    // **********************
    // Arguments: TrainingDataFile
    // Load the training file for DeSTIN
    

    // Data object containing source (training)
    DestinData DataSourceForTraining;

    DataSourceForTraining.LoadFile(argsStruc.strDestinTrainingFileName.c_str());
    int NumberOfUniqueLabels = DataSourceForTraining.GetNumberOfUniqueLabels();
    if ( NumberOfUniqueLabels==0 )
    {
        cout << "There seems to be something off with data source " << argsStruc.strDestinTrainingFileName.c_str() << endl;
        return 0;
    }

    // Data object of test source
    DestinData DataSourceForTesting;

    DataSourceForTesting.LoadFile((char*)(argsStruc.strTesting.c_str()));
    if ( DataSourceForTesting.GetNumberOfUniqueLabels()!=NumberOfUniqueLabels )
    {
        cout << "Test set does not have the same number of labels as train set " << endl;
        return 0;
    }

    // **************************
    // Preparing working data set
    // **************************
    // now get the data set creation parameters
   
    vector< pair<int,int> > vIndicesAndGTLabelToUse;


  

        // First part of code word RRRR = for time seeding
        string sNumInp = argsStruc.sCodeWord.substr(0,4);

        // if the first 4 digits are 0000 make a TRUE random, otherwise use the complete number.
        int iReserve = atoi( sNumInp.c_str() );
        if ( iReserve == 0 )
        {
            srand( time(NULL) );
        }
        else
        {
            srand( (unsigned int)argsStruc.seed );
        }

        // Second part of code word XX = number of inputs
        sNumInp = argsStruc.sCodeWord.substr(4,2);
        int NumberOfUniqueLabelsToUse = atoi( sNumInp.c_str() );

        // Last part of code word YYYYY
        sNumInp = argsStruc.sCodeWord.substr(6,5);
        int iNumberOfExamplesFromEachLabel=atoi( sNumInp.c_str() );

        // if iNumberOfExamplesFromEachLabel is 0 we randomly pick examples from the available
        // classes and only show them ONE TIME
        // Generate the examples from the dictates given here.
        vector< pair<int,int> > LabelsAndIndicesForUse;
        cout << "------------------" << endl;
        int DestinTrainSampleStep = 1;
        if(iNumberOfExamplesFromEachLabel == 0)
        {
            DestinTrainSampleStep = 25;
        }
        for(int iLabel=0;iLabel<NumberOfUniqueLabelsToUse;iLabel++)
        {
            int cnt = 0;
            vector<int> IndicesForThisLabel;
            DataSourceForTraining.GetIndicesForThisLabel(iLabel,IndicesForThisLabel);
            if ( IndicesForThisLabel.size() > iNumberOfExamplesFromEachLabel && iNumberOfExamplesFromEachLabel != 0)
            {
                for(int jj=0;jj<iNumberOfExamplesFromEachLabel;jj++)
                {
                    cnt++;
                    pair<int,int> P;
                    P.first = IndicesForThisLabel[jj];
                    P.second = iLabel;
                    LabelsAndIndicesForUse.push_back(P);
                }
            }
            else
            {
                for(int jj=0;jj<IndicesForThisLabel.size();jj=jj+DestinTrainSampleStep)
                {
                    cnt++;
                    pair<int,int> P;
                    P.first = IndicesForThisLabel[jj];
                    P.second = iLabel;
                    LabelsAndIndicesForUse.push_back(P);
                }

            }
            cout << "Label: " << iLabel << " got " << cnt << " unique sample(s)." << endl;
        }
        iNumberOfExamplesFromEachLabel = LabelsAndIndicesForUse.size()/NumberOfUniqueLabelsToUse;

        // Now generate MAX_CNT+1000 random numbers from 0 to LabelsAndIndicesForUse-1
        // and use these to populate vIndicesAndGTLabelToUse

        // Debug list of labels to be used
        int * Picked;
        Picked = (int *) malloc(sizeof(int) * NumberOfUniqueLabels);

        for(int jj=0;jj<NumberOfUniqueLabels;jj++)
        {
            Picked[jj]=0;
        }

        int Digit;
        int iChoice;
        for(int jj=0;jj<argsStruc.MAX_CNT;jj++)
        {
            //pick the digit first...
            Digit = rand() % NumberOfUniqueLabelsToUse;
            iChoice = Digit * iNumberOfExamplesFromEachLabel;
            iChoice = iChoice+rand() % iNumberOfExamplesFromEachLabel;

            pair<int,int> P;
            P = LabelsAndIndicesForUse[iChoice];

            vIndicesAndGTLabelToUse.push_back( P );
            // Debug counter of labels used by label
            Picked[P.second] += 1;
        }

        // Debug information on amount of examples we use each label
        cout << "------------------" << endl;
        for(int jj=0;jj<NumberOfUniqueLabels;jj++)
        {
            cout << "Label: " << jj << " will show " << Picked[jj] << " sample(s)." << endl;
        }
        free( Picked);
        cout << "------------------" << endl;




    // ***********************
    // Creating DeSTIN network
    // ***********************
    int SEQ_LENGTH = 0;
    int** SEQ;
    int* ImageInput;
    int NumberOfLayers;

    CreateDestinOnTheFly(argsStruc.ParametersFileName, NumberOfLayers, DKernel,
                          DataSourceForTraining, SEQ_LENGTH, SEQ, ImageInput);

    for (int i=0; i<NumberOfLayers;i++)
    {
        cout << "DeSTIN Layer information" << endl;
        cout << "Layer: " << DKernel[i].GetID() << endl;
        cout << "Dimension (row, col): " << DKernel[i].GetNumberOfRows() << " X " << DKernel[i].GetNumberOfCols() << endl;
        cout << "Input each node: " << DKernel[i].GetNumberOfInputDimensionlity() << endl;
        cout << "Centroids: " << DKernel[i].GetNumberOfStates() << endl;
        cout << endl;
    }

    cout << "------------------" << endl;
    cout << "Run Destin" << endl;
    cout << "Images to be processed: " << argsStruc.MAX_CNT << endl;
    cout << "Each image moves: " << SEQ_LENGTH << " times." << endl;

    double procces = 0.1;
    for(int i=0;i< argsStruc.MAX_CNT;i++)
    {
        if(i > (argsStruc.MAX_CNT-1)*procces)
        {
            cout << procces*100 << "%" << endl;
            procces+=0.1;
        }
        stringstream xml;
        xml << "<destin>" << endl;

        pair<int,int> element = vIndicesAndGTLabelToUse[i];
        int indexOfExample = element.first;
        int label = element.second;
		
        time_t iStart = time(NULL);
		
        for(int seq=0;seq<SEQ_LENGTH;seq++)
        {
            stringstream xmlLayer;
            // Run lowest layer (Kernel)

            time_t lStart = time(NULL);
			
            DataSourceForTraining.SetShiftedDeviceImage(indexOfExample, SEQ[seq][0], SEQ[seq][1], ImageInput[0], ImageInput[1]);
            DKernel[0].DoDestin(DataSourceForTraining.GetPointerDeviceImage(),&xmlLayer);
            //TODO: is the order of layer evaluation going in the right order?
            for(int l=1;l<NumberOfLayers;l++)
            {
                DKernel[l].DoDestin(DKernel[l-1].GetDevicePointerBeliefs(),&xmlLayer);
            }
            time_t lStop = time(NULL);
            xmlLayer << "<layerRuntime>" << lStop-lStart << "</layerRuntime>" << endl;
            if(seq == SEQ_LENGTH-1)
            {
                xml << xmlLayer.str().c_str();
            }
            xmlLayer.clear();
        }
        time_t iStop = time(NULL);
        xml << "<image id=\"" << i << "\" label=\"" << label << "\" labelIndex=\"" << indexOfExample << "\" runtime=\"" << iStop-iStart << "\" />" << endl;
        xml << "</destin>" << endl;
        if(i == argsStruc.MAX_CNT-1)
        {
            pugi::xml_document outputFile;
            outputFile.load(xml.str().c_str());
            string file = argsStruc.strDestinNetworkFileToRead;
            stringstream num;
            num << "-" << i;
            file.insert(file.length()-4, num.str());
            outputFile.save_file(file.c_str());
        }
    }
    time_t destinStop = time(NULL);
    cout << "Time run: " << destinStop-destinStart << endl;

    delete [] DKernel; 

    return 0;
}

// Simple run command: destinCuda(.exe) 00010100000 120 2:3 ./config.xml ../../data/MNISTTraining32 -D D
int main(int argc, char* argv[])
{
    // ********************
    // Startup check DeSTIN
    // ********************
    // There should be 8 or 9 arguments at this time if not show how to use DeSTIN
    
    DestinCuda dc;
    
    // arguments processing
    CommandArgsStuc argsStruc;
    if(dc.parseCommandArgs( argc, argv, argsStruc)!=0){
    	return 1;
    }
    
    
    if ( argc==8 || argc==9 )
    {
        cout << "Starting DeSTIN" << endl;
        cout << "------------------" << endl;
        
        return dc.MainDestinExperiments(argsStruc);
    }
    else
    {
        dc.PrintHelp();
        return 0;
    }
}
